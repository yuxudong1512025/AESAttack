#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"pch.h"

using namespace std;
#define  CHECK(call){\
	const hipError_t error = call;\
	if (error != hipSuccess) {\
		printf_s("Error: %s:%d, ", __FILE__, __LINE__);\
		printf_s("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(-10 * error);\
	}\
}\


__device__  inline u8 XTIME(u8 x) {
	return ((x << 1) ^ ((x & 0x80) ? 0x1b : 0x00));
}
__device__  inline u8 multiply(u8 a, u8 b) {
	u8 temp[8] = { a };
	u8 tempmultiply = 0x00;
	u32 i = 0;
	for (i = 1; i < 8; i++) {
		temp[i] = XTIME(temp[i - 1]);
	}
	tempmultiply = (b & 0x01) * a;
	for (i = 1; i <= 7; i++) {
		tempmultiply ^= (((b >> i) & 0x01) * temp[i]);
	}
	return tempmultiply;
}



__constant__ u8 sbox[256];

const u8 S_Box[256] = {
	 0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76 ,
	 0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0 ,
	 0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15 ,
	 0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75 ,
	 0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84 ,
	 0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF ,
	 0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8 ,
	 0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2 ,
	 0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73 ,
	 0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB ,
	 0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79 ,
	 0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08 ,
	 0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A ,
	 0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E ,
	 0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF ,
	 0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

/*����ciphertxt����֯��ʽ
	ciphertxt0=	00,07,10,17,20,27,....,ncountn-0,ncountn-7;
	temp=sbox[(0-256)^00]
	temp1=sbox[(0-256)^07]

*/

__global__ void uncode(u8 * cipher, u8 *record, int Countn, u8 info1, u8 info2) {//blocks=256,threads=256,cipher=2*countn��byte ���ϰ�����°��� 0,7/10,13
	u8 idx = threadIdx.x;
	u8 bdx = blockIdx.x;
	u32 it = (idx + bdx * blockDim.x);
	u8 temp, temp1;
	for (int i = 0; i < Countn; i ++ ) {
		temp = sbox[idx ^ cipher[2 * i]];
		temp1 = sbox[bdx ^ cipher[2 * i + 1]];
		record[i * Countn + it] = multiply(temp, info1) ^ multiply(temp1, info2);
	}
}

#define doublef(x) (x*x)
//sei��ʽ��д����ƽ���ͣ���


__global__ void  kernel(u8 *record0, u8 *record1, int Countn, float *maxSEI, u32 *maxKey) {//65536*256*256*1
	u32 idx = threadIdx.x;
	u32 right = (blockIdx.y*blockDim.x) + threadIdx.x;
	u32 left = blockIdx.x;
	u32 key = (left << 16) + right;
	__shared__ u32 partialMax[256];
	__shared__ u32 partialKey[256];
	u8 temp1, temp2;
	int Count[256];//256*4=1kb
	for (int i = 0; i < Countn; i++) {
		temp1 = record0[i*Countn + left];
		temp2 = record1[i*Countn + right];
		Count[(temp1^temp2)] += 1;
	}
	u32 temp = 0;
	for (int i = 0; i < 256; i += 32) {
		temp += doublef(Count[i]) + doublef(Count[i + 1]) + doublef(Count[i + 2]) + doublef(Count[i + 3]) + doublef(Count[i + 4]) + doublef(Count[i + 5]) + doublef(Count[i + 6]) + doublef(Count[i + 7]);
		temp += doublef(Count[i + 8]) + doublef(Count[i + 9]) + doublef(Count[i + 10]) + doublef(Count[i + 11]) + doublef(Count[i + 12]) + doublef(Count[i + 13]) + doublef(Count[i + 14]) + doublef(Count[i + 15]);
		temp += doublef(Count[i + 16]) + doublef(Count[i + 16]) + doublef(Count[i + 17]) + doublef(Count[i + 18]) + doublef(Count[i + 20]) + doublef(Count[i + 21]) + doublef(Count[i + 22]) + doublef(Count[i + 23]);
		temp += doublef(Count[i + 24]) + doublef(Count[i + 25]) + doublef(Count[i + 26]) + doublef(Count[i + 27]) + doublef(Count[i + 28]) + doublef(Count[i + 29]) + doublef(Count[i + 30]) + doublef(Count[i + 31]);

	}

	partialKey[idx] = key;
	partialMax[idx] = temp;
	//printf("%d %d key=%d ,sei=%f key=%d\n", idx, 1, key, sei, key);
	__syncthreads();

	for (int stride = blockDim.x / 2; stride > 0; stride = stride >> 1) {
		if (idx < stride&&partialMax[idx] < partialMax[idx + stride]) {
			partialMax[idx] = partialMax[idx + stride];
			partialKey[idx] = partialKey[idx + stride];
		}
		__syncthreads();
	}


	if (idx == 0) {
		*(maxSEI + blockIdx.y + blockDim.x*blockIdx.x) = partialMax[idx];
		*(maxKey + blockIdx.y + blockDim.x*blockIdx.x) = partialKey[idx];
	}

}

__host__ void randcipher(u8 *cipher, int Countn) {
	for (int i = 0; i < Countn * 4; i++)
		*(cipher + i) = rand() * 1000 % 256;
}
extern "C"
u32 getKey(u8*ciphertxt0, u8*ciphertxt1,int Countn) {
	//get device information
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("using device %d : %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	int roundn = 65536;
	int blocks = 1 << 8;
	int threads = 1 << 8;
	int btn = blocks * threads;
	int nu8 = btn * Countn * sizeof(u8);
	//	printf_s("Matrix size:nx %d ny %d\n", nx, ny);

	dim3 block(blocks, 1);
	dim3 grid(threads, 1);



	//printf_s("%d", nu8 * 2 + Countn * 4 + 16 * 16);

	u8 *cipher1,*cipher2;
	hipMalloc((void **)&cipher1, Countn * 2 * sizeof(u8));
	hipMalloc((void **)&cipher2, Countn * 2 * sizeof(u8));
	hipMemcpy(cipher1, ciphertxt0, Countn * 2 * sizeof(u8), hipMemcpyHostToDevice);
	hipMemcpy(cipher2, ciphertxt1, Countn * 2 * sizeof(u8), hipMemcpyHostToDevice);

	hipMemcpy(sbox, S_Box, 16 * 16 * sizeof(u8), hipMemcpyHostToDevice);

	u8 *Record0, *Record1;


	hipMalloc((void **)&Record0, nu8);
	hipMalloc((void **)&Record1, nu8);
	u8 mode0 = 0x0e, mode1 = 0x0b;
	uncode << <grid, block >> > (cipher1, Record0, Countn, mode0, mode1);
	CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����

	mode0 = 0x0d;  mode1 = 0x09;
	uncode << <grid, block >> > (cipher2, Record1, Countn, mode0, mode1);
	CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����

	dim3 block2(blocks, 1);
	dim3 grid2(roundn, blocks);

	nu8 = blocks * blocks*blocks * sizeof(u32);
	float *maxSEI;
	u32 *maxKey;


	hipMalloc((void **)&maxSEI, nu8);
	hipMalloc((void **)&maxKey, nu8);

	kernel << <grid2, block2 >> > (Record0, Record1, Countn, maxSEI, maxKey);


	CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����


	printf("success\n");
	hipFree(Record0);
	hipFree(Record1);
	hipFree(cipher1);
	hipFree(cipher2);
	hipFree(maxSEI);
	hipFree(maxKey);

	return 0;
}