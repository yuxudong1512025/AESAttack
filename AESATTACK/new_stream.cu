#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"pch.h"
#include"AES.h"

using namespace std;
#define  CHECK(call){\
	const hipError_t error = call;\
	if (error != hipSuccess) {\
		printf_s("Error: %s:%d, ", __FILE__, __LINE__);\
		printf_s("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(-10 * error);\
	}\
}\


__device__ inline u8 GFMul(u8 a, u8 b)
{
	u8 p = 0;
	u8 hi_bit_set;
	for (int counter = 0; counter < 8; counter++) {
		if ((b & u8(1)) != 0) {
			p ^= a;
		}
		hi_bit_set = (u8)(a & u8(0x80));
		a <<= 1;
		if (hi_bit_set != 0) {
			a ^= 0x1b; /* x^8 + x^4 + x^3 + x + 1 */
		}
		b >>= 1;
	}
	return p;
}





u8 Inv_S_Box[256] = {
	 0x52, 0x09, 0x6A, 0xD5, 0x30, 0x36, 0xA5, 0x38, 0xBF, 0x40, 0xA3, 0x9E, 0x81, 0xF3, 0xD7, 0xFB ,
	 0x7C, 0xE3, 0x39, 0x82, 0x9B, 0x2F, 0xFF, 0x87, 0x34, 0x8E, 0x43, 0x44, 0xC4, 0xDE, 0xE9, 0xCB ,
	 0x54, 0x7B, 0x94, 0x32, 0xA6, 0xC2, 0x23, 0x3D, 0xEE, 0x4C, 0x95, 0x0B, 0x42, 0xFA, 0xC3, 0x4E ,
	 0x08, 0x2E, 0xA1, 0x66, 0x28, 0xD9, 0x24, 0xB2, 0x76, 0x5B, 0xA2, 0x49, 0x6D, 0x8B, 0xD1, 0x25 ,
	 0x72, 0xF8, 0xF6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xD4, 0xA4, 0x5C, 0xCC, 0x5D, 0x65, 0xB6, 0x92 ,
	 0x6C, 0x70, 0x48, 0x50, 0xFD, 0xED, 0xB9, 0xDA, 0x5E, 0x15, 0x46, 0x57, 0xA7, 0x8D, 0x9D, 0x84 ,
	 0x90, 0xD8, 0xAB, 0x00, 0x8C, 0xBC, 0xD3, 0x0A, 0xF7, 0xE4, 0x58, 0x05, 0xB8, 0xB3, 0x45, 0x06 ,
	 0xD0, 0x2C, 0x1E, 0x8F, 0xCA, 0x3F, 0x0F, 0x02, 0xC1, 0xAF, 0xBD, 0x03, 0x01, 0x13, 0x8A, 0x6B ,
	 0x3A, 0x91, 0x11, 0x41, 0x4F, 0x67, 0xDC, 0xEA, 0x97, 0xF2, 0xCF, 0xCE, 0xF0, 0xB4, 0xE6, 0x73 ,
	 0x96, 0xAC, 0x74, 0x22, 0xE7, 0xAD, 0x35, 0x85, 0xE2, 0xF9, 0x37, 0xE8, 0x1C, 0x75, 0xDF, 0x6E ,
	 0x47, 0xF1, 0x1A, 0x71, 0x1D, 0x29, 0xC5, 0x89, 0x6F, 0xB7, 0x62, 0x0E, 0xAA, 0x18, 0xBE, 0x1B ,
	 0xFC, 0x56, 0x3E, 0x4B, 0xC6, 0xD2, 0x79, 0x20, 0x9A, 0xDB, 0xC0, 0xFE, 0x78, 0xCD, 0x5A, 0xF4 ,
	 0x1F, 0xDD, 0xA8, 0x33, 0x88, 0x07, 0xC7, 0x31, 0xB1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xEC, 0x5F ,
	 0x60, 0x51, 0x7F, 0xA9, 0x19, 0xB5, 0x4A, 0x0D, 0x2D, 0xE5, 0x7A, 0x9F, 0x93, 0xC9, 0x9C, 0xEF ,
	 0xA0, 0xE0, 0x3B, 0x4D, 0xAE, 0x2A, 0xF5, 0xB0, 0xC8, 0xEB, 0xBB, 0x3C, 0x83, 0x53, 0x99, 0x61 ,
	 0x17, 0x2B, 0x04, 0x7E, 0xBA, 0x77, 0xD6, 0x26, 0xE1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0C, 0x7D
};

/*����ciphertxt����֯��ʽ
	ciphertxt0=	00,07,10,17,20,27,....,ncountn-0,ncountn-7;
	temp=sbox[(0-256)^00]
	temp1=sbox[(0-256)^07]

*/

__global__ void uncode(u8* InvSbox, u8 * cipher, u8 *record, int Countn, u8 info1, u8 info2) {//blocks=256,threads=256,cipher=2*countn��byte ���ϰ�����°��� 0,7/10,13
	u8 idx = threadIdx.x;
	u8 bdx = blockIdx.x;
	u32 it = (idx * 256 + bdx);
	u8 temp, temp1;
	for (int i = 0; i < Countn; i++) {
		temp = InvSbox[idx ^ cipher[2 * i]];
		temp1 = InvSbox[bdx ^ cipher[2 * i + 1]];
		record[i * 65536 + it] = GFMul(temp, info1) ^ GFMul(temp1, info2);
	}
}

#define doublef(x) (u32)(x*x)
//sei��ʽ��д����ƽ���ͣ���

__global__ void getMaxSEI(u32 *maxSEI, u32 *maxKey, u32 *testkey,u32 *testSEI,int id) {//<<<(16384,1),(1024,1)>>>======<<<(256,1)(256,1)>>>
	const u32 tid = threadIdx.x;
	const u32 it = tid + blockIdx.x*blockDim.x;
	for (int stride = blockDim.x *gridDim.x / 2; stride > 0; stride = stride >> 1) {
		if (it + stride < stride * 2 && maxSEI[it] <= maxSEI[it + stride]) {
			maxSEI[it] = maxSEI[it + stride];
			maxKey[it] = maxKey[it + stride];
		}
		__syncthreads();
	}
	__syncthreads();

	if (it == 0) {
		testkey[id] = maxKey[it];
		testSEI[id] = maxSEI[it];
	}
}

__global__ void  kernel(u8 *record0, u8 *record1, int Countn, u32 *maxSEI, u32 *maxKey,u8 *Count, int id) {//65536*256*256*1
	u32 idx = threadIdx.x;
	u32 right = (blockIdx.x*blockDim.x) + threadIdx.x;
	u32 left = id;
	u32 key = (left << 16) + right;

	u8 temp1, temp2;
	for (int i = 0; i < 256; i++)Count[i] = 0;
	for (int i = 0; i < Countn; i++) {
		temp1 = record0[i * 65536 + left];
		temp2 = record1[i * 65536 + right];
		Count[(temp1^temp2) * 65536 + right ] ++;
	}
	u32 temp = 0;
	for (int i = 0; i < 256; i ++) {
		temp += doublef(Count[i * 65536 + right]);

	}
	maxKey[right] = key;
	maxSEI[right] = temp;
}



extern "C"
u32 getKey(u8*ciphertxt0, u8*ciphertxt1, int Countn, const u32 &trueKey) {

	FILE *fp = fopen("a.txt", "a+");
	//get device information
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf_s("using device %d : %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	int roundn = 65536;
	int blocks = 1 << 8;
	int threads = 1 << 8;
	int btn = blocks * threads;
	int nu8 = btn * Countn * sizeof(u8);
	//	printf_s("Matrix size:nx %d ny %d\n", nx, ny);

	dim3 block(blocks, 1);
	dim3 grid(threads, 1);
	///////////////////////////////////////////////////////
	u16 left = trueKey >> 16, right = (u16)trueKey;
	//fprintf(fp, "key=%x-%x\n", left, right);
	//for (int i = 0; i < Countn; i++) {
	//	fprintf(fp,"%x %x %x %x\n", ciphertxt0[2 * i], ciphertxt0[2 * i + 1], ciphertxt1[2 * i], ciphertxt1[2 * i + 1]);
	//}
	///////////////////////////////////////////////////////
	//printf_s("%d", nu8 * 2 + Countn * 4 + 16 * 16);

	u8 *cipher1, *cipher2, *InvSbox;

	CHECK(hipMalloc((void **)&InvSbox, 256 * sizeof(u8)));
	CHECK(hipMalloc((void **)&cipher1, Countn * 2 * sizeof(u8)));
	CHECK(hipMalloc((void **)&cipher2, Countn * 2 * sizeof(u8)));
	CHECK(hipMemcpy(cipher1, ciphertxt0, Countn * 2 * sizeof(u8), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(cipher2, ciphertxt1, Countn * 2 * sizeof(u8), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(InvSbox, Inv_S_Box, 256 * sizeof(u8), hipMemcpyHostToDevice));

	u8 *Record0, *Record1;


	CHECK(hipMalloc((void **)&Record0, nu8));
	CHECK(hipMalloc((void **)&Record1, nu8));
	u8 mode0 = 0x0e, mode1 = 0x0b;
	uncode << <grid, block >> > (InvSbox, cipher1, Record0, Countn, mode0, mode1);
	CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����

	mode0 = 0x0d;  mode1 = 0x09;
	uncode << <grid, block >> > (InvSbox, cipher2, Record1, Countn, mode0, mode1);
	CHECK(hipDeviceSynchronize());//���cuda�豸ͬ�����

	//u8 *hostRecord0, *hostRecord1;
	//hostRecord0 = (u8 *)malloc(nu8);
	//hostRecord1 = (u8 *)malloc(nu8);

	//CHECK(hipMemcpy(hostRecord0, Record0, nu8, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(hostRecord1, Record1, nu8, hipMemcpyDeviceToHost));


	//for (int j = 0; j < Countn; j++) {
	//	fprintf(fp,"%x-%x ", hostRecord0[j*65536+left], hostRecord1[j * 65536+right]);
	//	fprintf(fp, "%x ", hostRecord0[j * 65536 + left]^hostRecord1[j * 65536 + right]);
	//}fprintf(fp,"\n");




	dim3 block2(blocks, 1);
	dim3 grid2(blocks, 1);

	nu8 = blocks * blocks * sizeof(u32);
	u32 *maxSEI, *maxKey, *testSEI, *testkey;
	u8 *Count;

	CHECK(hipMalloc((void **)&maxSEI, nu8));
	CHECK(hipMalloc((void **)&maxKey, nu8));
	CHECK(hipMalloc((void **)&testSEI, nu8));
	CHECK(hipMalloc((void **)&testkey, nu8));
	CHECK(hipMalloc((void **)&Count, blocks* blocks* blocks * sizeof(u8)));

	for (int i = 0; i < 65536; i++) {
		CHECK(hipMemset(Count, 0, blocks* blocks* blocks * sizeof(u8)));
		kernel << <grid2, block2 >> > (Record0, Record1, Countn, maxSEI, maxKey,Count,i);
		getMaxSEI << <grid2, block2 >> > (maxSEI, maxKey, testkey, testSEI,i);
	}
	


	u32 *SEIlist = (u32 *)malloc(nu8);
	u32 *KEYlist = (u32 *)malloc(nu8);
	CHECK(hipMemcpy(SEIlist, testSEI, nu8, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(KEYlist, testkey, nu8, hipMemcpyDeviceToHost));


	u32 ans = 0, sei = 0;
	for (int i = 0; i < 65536; i++) {
		if (sei < SEIlist[i]) {
			sei = SEIlist[i];
			ans = KEYlist[i];
		}
	}
	printf("%d--%x\n", sei, ans);
	printf("success\n");
	CHECK(hipFree(Record0));
	CHECK(hipFree(Record1));
	CHECK(hipFree(cipher1));
	CHECK(hipFree(cipher2));
	CHECK(hipFree(maxSEI));
	CHECK(hipFree(maxKey));
	CHECK(hipFree(Count));
	CHECK(hipFree(testkey));
	CHECK(hipFree(testSEI));
	free(SEIlist);
	free(KEYlist);
	return ans;
}